/*
Name:Md Kamal Hossain Chowdhury
Email: mhchowdhury@crimson.ua.edu 
Course: CS 691
Homework #: 1

*/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define DIES 0
#define ALIVE 1
#define blockSize 256

const int TILE_DIM = 32;



/* function to measure time taken */
double gettime(void) {
  struct timeval tval;

  gettimeofday(&tval, NULL);

  return( (double)tval.tv_sec + (double)tval.tv_usec/1000000.0 );
}
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

void printarray(int *a, int M, int N, FILE *fp) {
  int i, j;
  for (i = 0; i < M+2; i++) {
    for (j = 0; j< N+2; j++)
      fprintf(fp, "%d ", a[i*(N+2) + j]);
    fprintf(fp, "\n");
  }
}

int check_array(int *a, int M, int N) {
  int value=0;
  for (int i = 1; i < M+1; i++)
    for (int j = 1; j< N+1; j++)
      value+= a[i*(N+2) + j];
  return value;
}
int compare_array(int *a,int *b, int M, int N) {
  int flag=1;
  for (int i = 1; i < M+1; i++)
    for (int j = 1; j< N+1; j++)
      if(a[i*(N+2) + j]!=b[i*(N+2) + j])
        {

          printf("Failed life[%d][%d]=%d h_life[%d][%d]=%d\n",i,j,a[i*(N+2) + j],i,j,b[i*(N+2) + j]);
          flag= 0;
          return flag;
        }
  return flag;
}


__global__
void compute_gpu_stride(int *life, int *temp, int M, int N) {
  // int  value;
  int index_x = blockIdx.x * blockDim.x + threadIdx.x+1;
  // int index_y = blockIdx.y * blockDim.y + threadIdx.y+1;
 
  int strid=blockDim.x*gridDim.x;
  
  
  int neighbors;
   
    for (int i = 1; i <N+1 ; i++){
        for(int j=index_x ;j<N+1; j+=strid){
         int id=i*(N+2)+j;
         neighbors = life[id + (N + 2)] +                           // Upper neighbor
                    life[id - (N + 2)] +                           // Lower neighbor
                    life[id + 1] +                                      // Right neighbor
                    life[id - 1] +                                      // Left neighbor
                    life[id + (N + 3)] + life[id - (N + 3)] + // Diagonal neighbors
                    life[id - (N + 1)] + life[id + (N + 1)];


        temp[id] = (neighbors == 3 || (neighbors == 2 && life[id]))? 1 : 0;
        }
        }

  
 
   }

__global__ 
void compute_shared_gpu(int *life, int *temp, int M, int N)
{
  int neighbors=0;
  


   
	int col = (blockDim.x - 2) * blockIdx.x + threadIdx.x;
	int row = (blockDim.y - 2) * blockIdx.y + threadIdx.y; 	

  int my_id= (row * (N+2) + col);
  int shared_id= (threadIdx.x * blockDim.y + threadIdx.y);
		
	int shared_size_x = blockDim.y;
	__shared__ int tile[TILE_DIM* TILE_DIM+1];
    //extern __shared__ TYPE sh_lattice[];

 	if (col < N+2 && row < N+2) {
        tile[shared_id] = life[my_id];
 	}
    __syncthreads();

    // CHECK IF
	/*if (col < size_i+neighs && row < size_j+neighs && 
		threadIdx.x >= (neighs-1) && threadIdx.x < blockDim.x-neighs && 
		threadIdx.y >= (neighs-1) && threadIdx.y < blockDim.y-neighs) {*/
    
    if (col < N+1 && row < N+1 && 
		threadIdx.x >= 1 && threadIdx.x < blockDim.x-1 && 
		threadIdx.y >= 1 && threadIdx.y < blockDim.y-1) {    
        
    //neighbors = neighbors_neighs(shared_id, shared_size_x-halo, sh_lattice, neighs, halo);	// decrease shared_size_x by 2 to use the same neighbors_neighs function than the rest of the implementations
    neighbors =  tile[shared_id - shared_size_x - 1];
    neighbors += tile[shared_id - shared_size_x];
    neighbors += tile[shared_id - shared_size_x + 1];
    neighbors += tile[shared_id - 1];
    neighbors += tile[shared_id + 1];
    neighbors += tile[shared_id + shared_size_x - 1];
    neighbors += tile[shared_id + shared_size_x];
    neighbors += tile[shared_id + shared_size_x + 1];

    temp[my_id] = (neighbors == 3 || (neighbors == 2 && life[my_id]))? 1 : 0;

    //check_rules(my_id, neighbors, d_lattice, d_lattice_new);
 	}
}
  
__global__
void compute_gpu(int *life, int *temp, int M, int N) {
  
     
  int x = blockIdx.x * blockDim.x + threadIdx.x+1;
  int y = blockIdx.y * blockDim.y + threadIdx.y+1;

  
  //int width = gridDim.x * TILE_DIM;   
  int id= x*(N+2)+y;
  int neighbors;
  //printf("gridDim=%d width=%d\n",gridDim,width);
  if(x<=N &&y<=N){
      neighbors = life[id + (N + 2)] +                           // Upper neighbor
                    life[id - (N + 2)] +                           // Lower neighbor
                    life[id + 1] +                                      // Right neighbor
                    life[id - 1] +                                      // Left neighbor
                    life[id + (N + 3)] + life[id - (N + 3)] + // Diagonal neighbors
                    life[id - (N + 1)] + life[id + (N + 1)];

        temp[id] = (neighbors == 3 || (neighbors == 2 && life[id]))? 1 : 0;
    }
    

}

   


void compute(int *life, int *temp, int M, int N) {
  int i, j, value;

  for (i = 1; i < M+1; i++) {
    for (j = 1; j < N+1; j++) {
      /* find out the value of the current cell */
      value = life[(i-1)*(N+2) + (j-1)] + life[(i-1)*(N+2) + j] + 
              life[(i-1)*(N+2) + (j+1)] + life[i*(N+2) + (j-1)] + 
              life[i*(N+2) + (j+1)] + life[(i+1)*(N+2) + (j-1)] + 
              life[(i+1)*(N+2) + j] + life[(i+1)*(N+2) + (j+1)] ;
     
      
      /* check if the cell dies or life is born */
      if (life[i*(N+2) + j]) { // cell was alive in the earlier iteration
	if (value < 2 || value > 3) {
	  temp[i*(N+2) + j] = DIES ;
	}
	else // value must be 2 or 3, so no need to check explicitly
	  temp[i*(N+2) + j] = ALIVE ; // no change
      } 
      else { // cell was dead in the earlier iteration
	if (value == 3) {
	  temp[i*(N+2) + j] = ALIVE;
	}
	else
	  temp[i*(N+2) + j] = DIES; // no change
      }
    }
  }

}


int main(int argc, char **argv) {
  int N, NTIMES, *life=NULL, *temp=NULL,*d_life=NULL,*d_temp=NULL,*h_life=NULL,*h_temp=NULL;
  int i, j, k;
  

  //int *life_stride=NULL,*temp_stride=NULL;
  // double t1_gpu,t2_gpu;
  
#if defined(DEBUG1) || defined(DEBUG2)
  FILE *fp;
  char filename[32];
#endif
if (argc != 4) {
        printf("Usage: %s <board size> <max number of generations> <directory for output file>\n", argv[0]);
        return -1;
    }
  N = atoi(argv[1]);
  NTIMES = atoi(argv[2]);
  char *directory = argv[3];

  /* Allocate memory for both arrays */
  life = (int *)malloc((N+2)*(N+2)*sizeof(int));
  temp = (int *)malloc((N+2)*(N+2)*sizeof(int));
  //life_stride = (int *)malloc((N+2)*(N+2)*sizeof(int));
  //temp_stride = (int *)malloc((N+2)*(N+2)*sizeof(int));

  /* Initialize the boundaries of the life matrix */
  for (i = 0; i < N+2; i++) {
    life[i*(N+2)] = life[i*(N+2) + (N+1)] = DIES ;
    temp[i*(N+2)] = temp[i*(N+2) + (N+1)] = DIES ;
  }
  for (j = 0; j < N+2; j++) {
    life[j] = life[(N+1)*(N+2) + j] = DIES ;
    temp[j] = temp[(N+1)*(N+2) + j] = DIES ;
  }

  /* Initialize the life matrix */
  for (i = 1; i < N+1; i++) {
    for (j = 1; j< N+1; j++) {
      srand(54321|i);
      if (drand48() < 0.5) 
	      life[i*(N+2) + j] = ALIVE ;
      else
	      life[i*(N+2) + j] = DIES ;
    }
  }
   //life_stride=life;  //copy life for stride kernel use
   //temp_stride=temp; //copy temp for stride kernel use

//gpu programming
  // int numBlocks = (N + blockSize - 1) / blockSize;
  // dim3 dimGrid_strid(numBlocks,numBlocks,1);
 
 
  //int blockSizeShared=(N+(TILE_DIM-2)-1)/(TILE_DIM-2);
  // dim3 dimGrid(blockSizeShared, blockSizeShared, 1);
  // dim3 dimBlock(TILE_DIM, TILE_DIM, 1);

  
  // dim3 dimGrid((N+2+(TILE_DIM-2)-1)/(TILE_DIM-2), (N+2+(TILE_DIM-2)-1)/(TILE_DIM-2), 1);
  // dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
  dim3 dimBlock(8, 8);
  dim3 dimGrid((N + dimBlock.x -1) / dimBlock.x, (N+dimBlock.y -1) / dimBlock.y);
  fprintf(stderr,"dimGrid: %d %d %d. dimBlock: %d %d %d\n",
         dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);
  
  hipMalloc(&d_life, (N+2)*(N+2)*sizeof(int)); 
  hipMalloc(&d_temp, (N+2)*(N+2)*sizeof(int));
  h_life = (int *)malloc((N+2)*(N+2)*sizeof(int));
  h_temp = (int *)malloc((N+2)*(N+2)*sizeof(int));
  
  hipMemcpy(d_life, life, (N+2)*(N+2)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_temp, temp, (N+2)*(N+2)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(h_life, d_life, (N+2)*(N+2)*sizeof(int), hipMemcpyDeviceToHost); 

  
#ifdef DEBUG1
  /* Display the initialized life matrix */
  fprintf(stderr,"Printing to file: gpuoutput.%d.0\n",N);
  sprintf(filename,"gpuoutput.%d.0",N);
  fp = fopen(filename, "w");
  printarray(life, N, N, fp);
  fprintf(fp,"\n-----------\n");
  printarray(h_life, N, N, fp);
  fclose(fp);
#endif
// events for timing
  hipEvent_t startEvent, stopEvent;
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  float ms;
  checkCuda( hipEventRecord(startEvent, 0) );

  for (k = 0; k < NTIMES; k += 2) {

    // compute_shared_gpu<<<dimGrid,dimBlock>>>(d_life,d_temp,N,N);
    // compute_shared_gpu<<<dimGrid,dimBlock>>>(d_temp,d_life,N,N);
    compute_gpu<<<dimGrid,dimBlock>>>(d_life,d_temp,N,N);
    compute_gpu<<<dimGrid,dimBlock>>>(d_temp,d_life,N,N);
  }

  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  fprintf(stderr,"----------------------------------\n");
  fprintf(stderr,"Baseline  GPU time is taken=%f ms for size=%d iterations=%d\n",ms,N,NTIMES);
    
  hipMemcpy(h_life, d_life, (N+2)*(N+2)*sizeof(int), hipMemcpyDeviceToHost); 
  hipMemcpy(h_temp, d_temp, (N+2)*(N+2)*sizeof(int), hipMemcpyDeviceToHost); 
  

#ifdef DEBUG1
  /* Display the life matrix after k iterations */
  printf("Printing to file: output.%d.%d\n",N,k);
  sprintf(filename,"gpuoutput.%d.%d",N,k);
  fp = fopen(filename, "w");
  printarray(h_temp, N, N, fp);
  fprintf(fp, "\n--------------------\n");
  printarray(h_life,N,N,fp);
  fclose(fp);
#endif

    FILE *f;
    f = fopen(directory, "w");
    if(f == NULL){
        printf("Error opening output file\n");
        return -1;
    }
    for(int i = 1; i < N+1; i++){
        for(int j = 1; j < N+1; j++){
            fprintf(f, " %d ", h_life[i*(N+2) + j]);
        }
        fprintf(f, "\n");
    }
    fclose(f);

  hipFree(d_life);
  hipFree(d_temp);
  free(h_life); 
  free(h_temp);
  free(life);
  free(temp);
  return 0;
}


